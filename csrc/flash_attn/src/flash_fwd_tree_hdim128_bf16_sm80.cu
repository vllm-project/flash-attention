// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"
#include "namespace_config.h"
#include "flash_fwd_tree_launch_template.h"

namespace FLASH_NAMESPACE {

template<>
void run_mha_fwd_tree_<cutlass::bfloat16_t, 128>(Flash_fwd_params_tree &params, hipStream_t stream) {
    run_mha_fwd_tree_hdim128<cutlass::bfloat16_t>(params, stream);
}

} // namespace FLASH_NAMESPACE